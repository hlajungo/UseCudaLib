#include "UseCuda.h"


Use_cuda::Use_cuda() : has_gpu(false)
{
  if (check_has_gpu())
  {
    set_has_gpu(true);
  }
}

bool Use_cuda::get_has_gpu()
{
  return has_gpu;
}

void Use_cuda::set_has_gpu(bool val)
{
  has_gpu = val;
}

bool Use_cuda::check_has_gpu()
{
  int device_count = 0;
  hipError_t err = hipGetDeviceCount(&device_count);

  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    return false;
  }

  // update has_gpu
  if (device_count > 0)
  {
    has_gpu = true;
  }
  else
  {
    has_gpu = false;
  }

  return device_count > 0;
}
